#include "hip/hip_runtime.h"
// Copyright (c) Meta Platforms, Inc. and affiliates.
// All rights reserved.
//
// This source code is licensed under the BSD-style license found in the
// LICENSE file in the root directory of this source tree.

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>
#include <vector>

// Smallest problems - use shared memory
template <typename scalar_t, int BLOCK_SIZE, int FEATURES_PER_THREAD>
__global__ void
fast_permute_small_kernel(const scalar_t *__restrict__ input,
                          const int64_t *__restrict__ permute_indices,
                          scalar_t *__restrict__ output, int64_t num_indices,
                          int64_t feature_size) {
  // smem buffer for features of a single token
  extern __shared__ char shared_mem[];
  scalar_t *shared_features = reinterpret_cast<scalar_t *>(shared_mem);

  // each block will handle a single token
  const int token_idx = blockIdx.x;
  const int thread_idx = threadIdx.x;

  // if in bounds (effectively M)
  if (token_idx < num_indices) {
    // load source token
    const int64_t src_idx = permute_indices[token_idx];

    // each thread loads features into smem
    for (int i = thread_idx; i < feature_size; i += BLOCK_SIZE) {
      shared_features[i] = input[src_idx * feature_size + i];
    }

    // wait for everyone to load...
    __syncthreads();

    // each thread writes features to output
    for (int i = thread_idx; i < feature_size; i += BLOCK_SIZE) {
      output[token_idx * feature_size + i] = shared_features[i];
    }
  }
}

// medium kernel - multiple tokens per block
template <typename scalar_t, int TOKENS_PER_BLOCK, int THREADS_PER_BLOCK>
__global__ void
fast_permute_medium_kernel(const scalar_t *__restrict__ input,
                           const int64_t *__restrict__ permute_indices,
                           scalar_t *__restrict__ output, int64_t num_indices,
                           int64_t feature_size) {
  // Here each block handles multiple tokens
  const int token_start = blockIdx.x * TOKENS_PER_BLOCK;
  const int thread_idx = threadIdx.x;

  // threads per token
  const int threads_per_token = THREADS_PER_BLOCK / TOKENS_PER_BLOCK;
  // token and thread offset
  const int local_token_idx = thread_idx / threads_per_token;
  const int token_thread_idx = thread_idx % threads_per_token;

  // Global token index
  const int token_idx = token_start + local_token_idx;

  if (token_idx < num_indices) {
    // load source index
    const int64_t src_idx = permute_indices[token_idx];

    // just like in small, each thread copies subset of features
    for (int i = token_thread_idx; i < feature_size; i += threads_per_token) {
      output[token_idx * feature_size + i] = input[src_idx * feature_size + i];
    }
  }
}

// large kernel - each block processes multiple rows and columns
template <typename scalar_t>
__global__ void
fast_permute_large_kernel(const scalar_t *__restrict__ input,
                          const int64_t *__restrict__ permute_indices,
                          scalar_t *__restrict__ output, int64_t num_indices,
                          int64_t feature_size, int batch_size) {}
